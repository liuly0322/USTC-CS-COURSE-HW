
#include <hip/hip_runtime.h>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <ctime>

// #define VERIFY
#define BLOCK 32 // 宏定义分块大小

__global__ void gemm_block(const float *A, const float *B, float *C, int N);
void gemm_verify(const float *A, const float *B, float *C, int N);

constexpr float rand_max = static_cast<float>(RAND_MAX);

int main(int argc, char *argv[]) {
    int N = 1 << 10;
    if (argc > 1)
        N = std::atoi(argv[1]);

    float *A;
    float *B;
    float *C;
    // Allocate memory for A, B, C
    auto size = N * N * sizeof(float);
    hipMallocManaged(&A, size);
    hipMallocManaged(&B, size);
    hipMallocManaged(&C, size);

    // Random initialize A, B
    for (int i = 0; i < N * N; i++) {
        A[i] = static_cast<float>(rand()) / rand_max;
        B[i] = static_cast<float>(rand()) / rand_max;
    }

    // Define grid size and block size
    const dim3 blockSize(BLOCK, BLOCK);
    const dim3 gridSize((N + BLOCK - 1) / BLOCK, (N + BLOCK - 1) / BLOCK);

    // Create CUDA events for timing
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record start event
    hipEventRecord(start);

    // Compute
    gemm_block<<<gridSize, blockSize>>>(A, B, C, N);

    // Record stop event
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Compute elapsed time
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Elapsed time: %f ms\n", elapsedTime);

    // Destroy CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);

#ifdef VERIFY
    // Verify the result
    gemm_verify(A, B, C, N);
#endif

    // Free memory
    hipFree(A);
    hipFree(B);
    hipFree(C);

    return 0;
}

/**
 * @brief gemm_block
 * one thread block share one block of A and B.
 * one thread compute one element of C.
 * @param A pointer to the first matrix
 * @param B pointer to the second matrix
 * @param C pointer to the result matrix
 * @param N the size of matrix
 * @return void
 */
__global__ void gemm_block(const float *A, const float *B, float *C, int N) {
    __shared__ float sA[BLOCK][BLOCK];
    __shared__ float sB[BLOCK][BLOCK];

    auto tx = threadIdx.x;
    auto ty = threadIdx.y;

    auto row = blockIdx.y * BLOCK + ty;
    auto col = blockIdx.x * BLOCK + tx;

    if (row >= N || col >= N)
        return;

    auto sum = 0.0F;

    for (auto m = 0; m < (N + BLOCK - 1) / BLOCK; m++) {
        sA[ty][tx] = A[row * N + m * BLOCK + tx];
        sB[ty][tx] = B[col + (m * BLOCK + ty) * N];
        __syncthreads();

        for (auto k = 0; k < BLOCK; k++)
            sum += sA[ty][k] * sB[k][tx];
        __syncthreads();
    }

    C[row * N + col] = sum;
}

void gemm_verify(const float *A, const float *B, float *C, int N) {
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            auto sum = 0.0F;
            for (int k = 0; k < N; k++)
                sum += A[i * N + k] * B[k * N + j];
            if (std::fabs(C[i * N + j] - sum) > 1e-4) {
                printf("C[%d][%d] = %f, sum = %f\n", i, j, C[i * N + j], sum);
                printf("Verification failed.\n");
                return;
            }
        }
    }
    printf("Verification succeeded.\n");
}
