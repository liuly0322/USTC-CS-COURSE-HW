
#include <hip/hip_runtime.h>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <ctime>
// #define VERIFY

__global__ void gemm_baseline(const float *A, const float *B, float *C, int N);
void gemm_verify(const float *A, const float *B, float *C, int N);

constexpr float rand_max = static_cast<float>(RAND_MAX);

int main(int argc, char *argv[]) {
    int N = 1 << 10;
    int block_size = 16;
    if (argc > 1)
        N = std::atoi(argv[1]);
    if (argc > 2)
        block_size = std::atoi(argv[2]);

    float *A;
    float *B;
    float *C;
    // Allocate memory for A, B, C
    auto size = N * N * sizeof(float);
    hipMallocManaged(&A, size);
    hipMallocManaged(&B, size);
    hipMallocManaged(&C, size);

    // Random initialize A, B
    for (int i = 0; i < N * N; i++) {
        A[i] = static_cast<float>(rand()) / rand_max;
        B[i] = static_cast<float>(rand()) / rand_max;
    }

    // Define grid size and block size
    const dim3 blockSize(block_size, block_size);
    const dim3 gridSize((N + block_size - 1) / block_size,
                        (N + block_size - 1) / block_size);

    // Create CUDA events for timing
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record start event
    hipEventRecord(start);

    // Compute
    gemm_baseline<<<gridSize, blockSize>>>(A, B, C, N);

    // Record stop event
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Compute elapsed time
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Elapsed time: %f ms\n", elapsedTime);

    // Destroy CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);

#ifdef VERIFY
    // Verify the result
    gemm_verify(A, B, C, N);
#endif

    // Free memory
    hipFree(A);
    hipFree(B);
    hipFree(C);

    return 0;
}

__global__ void gemm_baseline(const float *A, const float *B, float *C, int N) {
    auto row = blockIdx.y * blockDim.y + threadIdx.y;
    auto col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= N || col >= N)
        return;

    auto sum = 0.0F;
    for (int k = 0; k < N; k++)
        sum += A[row * N + k] * B[k * N + col];

    C[row * N + col] = sum;
}

void gemm_verify(const float *A, const float *B, float *C, int N) {
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            auto sum = 0.0F;
            for (int k = 0; k < N; k++)
                sum += A[i * N + k] * B[k * N + j];
            if (std::fabs(C[i * N + j] - sum) > 1e-4) {
                printf("C[%d][%d] = %f, sum = %f\n", i, j, C[i * N + j], sum);
                printf("Verification failed.\n");
                return;
            }
        }
    }
    printf("Verification succeeded.\n");
}
